/*
 * First version of Striped Arrayed Ungapped Alignment ( Not optimized )
 * 2021/01/06/13:17
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>

using namespace std;

int getCodonTableIndex(char aa){
    switch (aa) {
        case 'A':
            return 0;
        case 'R':
            return 1;
        case 'N':
            return 2;
        case 'D':
            return 3;
        case 'C':
            return 4;
        case 'Q':
            return 5;
        case 'E':
            return 6;
        case 'G':
            return 7;
        case 'H':
            return 8;
        case 'I':
            return 9;
        case 'L':
            return 10;
        case 'K':
            return 11;
        case 'M':
            return 12;
        case 'F':
            return 13;
        case 'P':
            return 14;
        case 'S':
            return 15;
        case 'T':
            return 16;
        case 'W':
            return 17;
        case 'Y':
            return 18;
        case 'V':
            return 19;
        case 'B':
            return 20;
        case 'Z':
            return 21;
        case 'X':
            return 22;
        case '*': default:
            return 23;
    }
}
int getSubMatrix(char targetAA, char queryAA){
    int targetIdx = getCodonTableIndex(targetAA);
    int queryIdx = getCodonTableIndex(queryAA);
    int largerIdx = (targetIdx > queryIdx) ? targetIdx : queryIdx;
    int smallerIdx = (targetIdx > queryIdx) ? queryIdx : targetIdx;

    static int codon_table[50][50]={
            { 4,-1,-2,-2, 0,-1,-1, 0,-2,-1,-1,-1,-1,-2,-1, 1, 0,-3,-2, 0,-2,-1,-1,-1,-4},
            {-1, 5, 0,-2,-3, 1, 0,-2, 0,-3,-2, 2,-1,-3,-2,-1,-1,-3,-2,-3,-1,-2, 0,-1,-4},
            {-2, 0, 6, 1,-3, 0, 0, 0, 1,-3,-3, 0,-2,-3,-2, 1, 0,-4,-2,-3, 4,-3, 0,-1,-4},
            {-2,-2, 1, 6,-3, 0, 2,-1,-1,-3,-4,-1,-3,-3,-1, 0,-1,-4,-3,-3, 4,-3, 1,-1,-4},
            { 0,-3,-3,-3, 9,-3,-4,-3,-3,-1,-1,-3,-1,-2,-3,-1,-1,-2,-2,-1,-3,-1,-3,-1,-4},
            {-1, 1, 0, 0,-3, 5, 2,-2, 0,-3,-2, 1, 0,-3,-1, 0,-1,-2,-1,-2, 0,-2, 4,-1,-4},
            {-1, 0, 0, 2,-4, 2, 5,-2, 0,-3,-3, 1,-2,-3,-1, 0,-1,-3,-2,-2, 1,-3, 4,-1,-4},
            { 0,-2, 0,-1,-3,-2,-2, 6,-2,-4,-4,-2,-3,-3,-2, 0,-2,-2,-3,-3,-1,-4,-2,-1,-4},
            {-2, 0, 1,-1,-3,-0, 0,-2, 8,-3,-3,-1,-2,-1,-2,-1,-2,-2, 2,-3, 0,-3, 0,-1,-4},
            {-1,-3,-3,-3,-1,-3,-3,-4,-3, 4, 2,-3, 1, 0,-3,-2,-1,-3,-1, 3,-3, 3,-3,-1,-4},
            {-1,-2,-3,-4,-1,-2,-3,-4,-3, 2, 4,-2, 2, 0,-3,-2,-1,-2,-1, 1,-4, 3,-3,-1,-4},
            {-1, 2, 0,-1,-3, 1, 1,-2,-1,-3,-2, 5,-1,-3,-1, 0,-1,-3,-2,-2, 0,-3, 1,-1,-4},
            {-1,-1,-2,-3,-1, 0,-2,-3,-2, 1, 2,-1, 5, 0,-2,-1,-1,-1,-1, 1,-3, 2,-1,-1,-4},
            {-2,-3,-3,-3,-2,-3,-3,-3,-1, 0, 0,-3, 0, 6,-4,-2,-2, 1, 3,-1,-3, 0,-3,-1,-4},
            {-1,-2,-2,-1,-3,-1,-1,-2,-2,-3,-3,-1,-2,-4, 7,-1,-1,-4,-3,-2,-2,-3,-1,-1,-4},
            { 1,-1, 1, 0,-1, 0, 0, 0,-1,-2,-2, 0,-1,-2,-1, 4, 1,-3,-2,-2, 0,-2, 0,-1,-4},
            { 0,-1, 0,-1,-1,-1,-1,-2,-2,-1,-1,-1,-1,-2,-1, 1, 5,-2,-2, 0,-1,-1,-1,-1,-4},
            {-3,-3,-4,-4,-2,-2,-3,-2,-2,-3,-2,-3,-1, 1,-4,-3,-2,11, 2,-3,-4,-2,-2,-1,-4},
            {-2,-2,-2,-3,-2,-1,-2,-3, 2,-1,-1,-2,-1, 3,-3,-2,-2, 2, 7,-1,-3,-1,-2,-1,-4},
            { 0,-3,-3,-3,-1,-2,-2,-3,-3, 3, 1,-2, 1,-1,-2,-2, 0,-3,-1, 4,-3, 2,-2,-1,-4},
            {-2,-1, 4, 4,-3, 0, 1,-1, 0,-3,-4, 0,-3,-3,-2, 0,-1,-4,-3,-3, 4,-3, 0,-1,-4},
            {-1,-2,-3,-3,-1,-2,-3,-4,-3, 3, 3,-3, 2, 0,-3,-2,-1,-2,-1, 2,-3, 3,-3,-1,-4},
            {-1, 0, 0, 1,-3, 4, 4,-2, 0,-3,-3, 1,-1,-3,-1, 0,-1,-2,-2,-2, 0,-3, 4,-1,-4},
            {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-4},
            {-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4, 1}
    };

    return codon_table[smallerIdx][largerIdx];
}
__device__
int getCodonTableIndexD(char aa){
    switch (aa) {
        case 'A':
            return 0;
        case 'R':
            return 1;
        case 'N':
            return 2;
        case 'D':
            return 3;
        case 'C':
            return 4;
        case 'Q':
            return 5;
        case 'E':
            return 6;
        case 'G':
            return 7;
        case 'H':
            return 8;
        case 'I':
            return 9;
        case 'L':
            return 10;
        case 'K':
            return 11;
        case 'M':
            return 12;
        case 'F':
            return 13;
        case 'P':
            return 14;
        case 'S':
            return 15;
        case 'T':
            return 16;
        case 'W':
            return 17;
        case 'Y':
            return 18;
        case 'V':
            return 19;
        case 'B':
            return 20;
        case 'Z':
            return 21;
        case 'X':
            return 22;
        case '*': default:
            return 23;
    }
}
__device__
int getSubMatrixD(char targetAA, char queryAA){
    int targetIdx = getCodonTableIndexD(targetAA);
    int queryIdx = getCodonTableIndexD(queryAA);
    int largerIdx = (targetIdx > queryIdx) ? targetIdx : queryIdx;
    int smallerIdx = (targetIdx > queryIdx) ? queryIdx : targetIdx;

    static int codon_table[50][50]={
            { 4,-1,-2,-2, 0,-1,-1, 0,-2,-1,-1,-1,-1,-2,-1, 1, 0,-3,-2, 0,-2,-1,-1,-1,-4},
            {-1, 5, 0,-2,-3, 1, 0,-2, 0,-3,-2, 2,-1,-3,-2,-1,-1,-3,-2,-3,-1,-2, 0,-1,-4},
            {-2, 0, 6, 1,-3, 0, 0, 0, 1,-3,-3, 0,-2,-3,-2, 1, 0,-4,-2,-3, 4,-3, 0,-1,-4},
            {-2,-2, 1, 6,-3, 0, 2,-1,-1,-3,-4,-1,-3,-3,-1, 0,-1,-4,-3,-3, 4,-3, 1,-1,-4},
            { 0,-3,-3,-3, 9,-3,-4,-3,-3,-1,-1,-3,-1,-2,-3,-1,-1,-2,-2,-1,-3,-1,-3,-1,-4},
            {-1, 1, 0, 0,-3, 5, 2,-2, 0,-3,-2, 1, 0,-3,-1, 0,-1,-2,-1,-2, 0,-2, 4,-1,-4},
            {-1, 0, 0, 2,-4, 2, 5,-2, 0,-3,-3, 1,-2,-3,-1, 0,-1,-3,-2,-2, 1,-3, 4,-1,-4},
            { 0,-2, 0,-1,-3,-2,-2, 6,-2,-4,-4,-2,-3,-3,-2, 0,-2,-2,-3,-3,-1,-4,-2,-1,-4},
            {-2, 0, 1,-1,-3,-0, 0,-2, 8,-3,-3,-1,-2,-1,-2,-1,-2,-2, 2,-3, 0,-3, 0,-1,-4},
            {-1,-3,-3,-3,-1,-3,-3,-4,-3, 4, 2,-3, 1, 0,-3,-2,-1,-3,-1, 3,-3, 3,-3,-1,-4},
            {-1,-2,-3,-4,-1,-2,-3,-4,-3, 2, 4,-2, 2, 0,-3,-2,-1,-2,-1, 1,-4, 3,-3,-1,-4},
            {-1, 2, 0,-1,-3, 1, 1,-2,-1,-3,-2, 5,-1,-3,-1, 0,-1,-3,-2,-2, 0,-3, 1,-1,-4},
            {-1,-1,-2,-3,-1, 0,-2,-3,-2, 1, 2,-1, 5, 0,-2,-1,-1,-1,-1, 1,-3, 2,-1,-1,-4},
            {-2,-3,-3,-3,-2,-3,-3,-3,-1, 0, 0,-3, 0, 6,-4,-2,-2, 1, 3,-1,-3, 0,-3,-1,-4},
            {-1,-2,-2,-1,-3,-1,-1,-2,-2,-3,-3,-1,-2,-4, 7,-1,-1,-4,-3,-2,-2,-3,-1,-1,-4},
            { 1,-1, 1, 0,-1, 0, 0, 0,-1,-2,-2, 0,-1,-2,-1, 4, 1,-3,-2,-2, 0,-2, 0,-1,-4},
            { 0,-1, 0,-1,-1,-1,-1,-2,-2,-1,-1,-1,-1,-2,-1, 1, 5,-2,-2, 0,-1,-1,-1,-1,-4},
            {-3,-3,-4,-4,-2,-2,-3,-2,-2,-3,-2,-3,-1, 1,-4,-3,-2,11, 2,-3,-4,-2,-2,-1,-4},
            {-2,-2,-2,-3,-2,-1,-2,-3, 2,-1,-1,-2,-1, 3,-3,-2,-2, 2, 7,-1,-3,-1,-2,-1,-4},
            { 0,-3,-3,-3,-1,-2,-2,-3,-3, 3, 1,-2, 1,-1,-2,-2, 0,-3,-1, 4,-3, 2,-2,-1,-4},
            {-2,-1, 4, 4,-3, 0, 1,-1, 0,-3,-4, 0,-3,-3,-2, 0,-1,-4,-3,-3, 4,-3, 0,-1,-4},
            {-1,-2,-3,-3,-1,-2,-3,-4,-3, 3, 3,-3, 2, 0,-3,-2,-1,-2,-1, 2,-3, 3,-3,-1,-4},
            {-1, 0, 0, 1,-3, 4, 4,-2, 0,-3,-3, 1,-1,-3,-1, 0,-1,-2,-2,-2, 0,-3, 4,-1,-4},
            {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-4},
            {-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4,-4, 1}
    };

    return codon_table[smallerIdx][largerIdx];
}

void fastaParser(string FilePath, string& parsedSeq){
    ifstream fastaStream(FilePath);
    string line;
    getline(fastaStream, line); // pass the first line.
    while(getline(fastaStream, line)){
        parsedSeq.append(line);
    }
    cout << "[" << parsedSeq << "]" << endl;
}

__global__ void unGappedAlignGPU_stripedArr(char* vectorCurr, char* vectorPrev, char queryi, char* target, char* max, int targetLeng)
{
    int i = threadIdx.x;
    int jump = blockDim.x;


    if(i == 0){
        char subVal = getSubMatrixD(queryi, target[0]);
        *vectorCurr = subVal > 0 ? subVal : 0;
        max[i] = (*vectorCurr > max[i]) ? *vectorCurr : max[i];
    }


    for (int j = i+1; j < targetLeng; j+= jump) {
        // < PARALLELIZE > //
        char tempA = *(vectorPrev + j - 1) + getSubMatrixD(queryi, target[j]);
        *(vectorCurr + j) = (tempA > 0) ? tempA : 0;
        max[j] = (tempA > max[j]) ? tempA : max[j];
    }
}

void unGappedAlignCPU(char* vectorCurr, char* vectorPrev, char queryi, char* target, char* max, int targetLeng)
{
    *vectorCurr = std::max(getSubMatrix(queryi, target[0]), 0);
    *max = std::max(*vectorCurr, *max);
    for (int j = 1; j < targetLeng; j++) {
        *(vectorCurr + j) = std::max(*(vectorPrev + j - 1) + getSubMatrix(queryi, target[j]), 0);
        *max = std::max(*(vectorCurr + j), *max);
    }
}

#define HUMANHEMALPHA5 "MVLSPADKTNVKAAWGKVGAHAGEYGAEALERMFLSFPTTKTYFPHFDLSHGSAQVKGHGKKVADALTNAVAHVDDMPNALSALSDLHAHKLRVDPVNFKLLSHCLLVTLAAHLPAEFTPAVHASLDKFLASVSTVLTSKYRMVLSPADKTNVKAAWGKVGAHAGEYGAEALERMFLSFPTTKTYFPHFDLSHGSAQVKGHGKKVADALTNAVAHVDDMPNALSALSDLHAHKLRVDPVNFKLLSHCLLVTLAAHLPAEFTPAVHASLDKFLASVSTVLTSKYRMVLSPADKTNVKAAWGKVGAHAGEYGAEALERMFLSFPTTKTYFPHFDLSHGSAQVKGHGKKVADALTNAVAHVDDMPNALSALSDLHAHKLRVDPVNFKLLSHCLLVTLAAHLPAEFTPAVHASLDKFLASVSTVLTSKYRMVLSPADKTNVKAAWGKVGAHAGEYGAEALERMFLSFPTTKTYFPHFDLSHGSAQVKGHGKKVADALTNAVAHVDDMPNALSALSDLHAHKLRVDPVNFKLLSHCLLVTLAAHLPAEFTPAVHASLDKFLASVSTVLTSKYRMVLSPADKTNVKAAWGKVGAHAGEYGAEALERMFLSFPTTKTYFPHFDLSHGSAQVKGHGKKVADALTNAVAHVDDMPNALSALSDLHAHKLRVDPVNFKLLSHCLLVTLAAHLPAEFTPAVHASLDKFLASVSTVLTSKYR"
#define HUMANHEMBETA5  "MVHLTPEEKSAVTALWGKVNVDEVGGEALGRLLVVYPWTQRFFESFGDLSTPDAVMGNPKVKAHGKKVLGAFSDGLAHLDNLKGTFATLSELHCDKLHVDPENFRLLGNVLVCVLAHHFGKEFTPPVQAAYQKVVAGVANALAHKYHMVHLTPEEKSAVTALWGKVNVDEVGGEALGRLLVVYPWTQRFFESFGDLSTPDAVMGNPKVKAHGKKVLGAFSDGLAHLDNLKGTFATLSELHCDKLHVDPENFRLLGNVLVCVLAHHFGKEFTPPVQAAYQKVVAGVANALAHKYHMVHLTPEEKSAVTALWGKVNVDEVGGEALGRLLVVYPWTQRFFESFGDLSTPDAVMGNPKVKAHGKKVLGAFSDGLAHLDNLKGTFATLSELHCDKLHVDPENFRLLGNVLVCVLAHHFGKEFTPPVQAAYQKVVAGVANALAHKYHMVHLTPEEKSAVTALWGKVNVDEVGGEALGRLLVVYPWTQRFFESFGDLSTPDAVMGNPKVKAHGKKVLGAFSDGLAHLDNLKGTFATLSELHCDKLHVDPENFRLLGNVLVCVLAHHFGKEFTPPVQAAYQKVVAGVANALAHKYHMVHLTPEEKSAVTALWGKVNVDEVGGEALGRLLVVYPWTQRFFESFGDLSTPDAVMGNPKVKAHGKKVLGAFSDGLAHLDNLKGTFATLSELHCDKLHVDPENFRLLGNVLVCVLAHHFGKEFTPPVQAAYQKVVAGVANALAHKYH"
#define HUMANHEMALPHA "MVLSPADKTNVKAAWGKVGAHAGEYGAEALERMFLSFPTTKTYFPHFDLSHGSAQVKGHGKKVADALTNAVAHVDDMPNALSALSDLHAHKLRVDPVNFKLLSHCLLVTLAAHLPAEFTPAVHASLDKFLASVSTVLTSKYR"
#define HUMANHEMBETA "MVHLTPEEKSAVTALWGKVNVDEVGGEALGRLLVVYPWTQRFFESFGDLSTPDAVMGNPKVKAHGKKVLGAFSDGLAHLDNLKGTFATLSELHCDKLHVDPENFRLLGNVLVCVLAHHFGKEFTPPVQAAYQKVVAGVANALAHKYH"

#define HUMANHEMALPHA0 "MVLSPADKTNVKA"
#define HUMANHEMBETA0  "MVHLTPEEKSA"

#define QUERYSEQUENCE HUMANHEMALPHA5
#define TARGETSEQUENCE HUMANHEMBETA5
int main() {
    using namespace std;
    string hemA, hemB;

    int iter = 100;
    // CPU task.
    double CPUtime, GPUtime;
    {
        char humanHemAlpha[] = QUERYSEQUENCE;
        char humanHemBeta[] = TARGETSEQUENCE;

        char *query = humanHemAlpha;
        char *target = humanHemBeta;

        int targetSize = strlen(target);
        printf("Start : with target size %d \n", targetSize);

        const clock_t begin_time = clock();
        char max = 0;

        char *vectorCurr = (char *) malloc(targetSize * sizeof(char));
        char *vectorPrev = (char *) malloc(targetSize * sizeof(char));
        memset(vectorPrev, 0, targetSize * sizeof(char));


        for (int tx = 0; tx < iter; tx++) {
            for (int i = 0; query[i] != '\0'; i++) {
                unGappedAlignCPU(vectorCurr, vectorPrev, query[i], target, &max, targetSize);

                char *vectorTmp = vectorPrev;
                vectorPrev = vectorCurr;
                vectorCurr = vectorTmp;
            }
        }
        CPUtime = clock() - begin_time;
        cout << "CPU result : " << (int) max << endl;
        cout << "Elapsed time : " << CPUtime / CLOCKS_PER_SEC << endl;
    }
    //GPU task with arrayed striped
    {
        int NUM_CUDA_THREAD = 256;

        char humanHemAlpha[] = QUERYSEQUENCE;
        char humanHemBeta[] = TARGETSEQUENCE;

        int querySize = strlen(humanHemAlpha);
        int targetSize = strlen(humanHemBeta);

        char *query, *target;
        query = humanHemAlpha;

        const clock_t begin_time = clock();

        hipMalloc((void **) &target, targetSize * sizeof(char));
        hipMemcpy(target, humanHemBeta, targetSize, hipMemcpyHostToDevice);

        char *vectorCurr, *vectorPrev;
        hipMalloc((void **) &vectorCurr, targetSize * sizeof(char));
        hipMalloc((void **) &vectorPrev, targetSize * sizeof(char));
        hipMemset(vectorCurr, 0, targetSize*sizeof(char));
        hipMemset(vectorPrev, 0, targetSize*sizeof(char));

        char *max;
        hipMalloc((void **) &max, NUM_CUDA_THREAD * sizeof(char));
        hipMemset(max, 0, NUM_CUDA_THREAD*sizeof(char));

        char maxRes = 0;
        char *maxHost = (char*)malloc(NUM_CUDA_THREAD * sizeof(char));
        for (int tx = 0; tx < iter; tx++) {
            for(int i=0; i<querySize; i++){
                hipMemset(vectorCurr, 0, sizeof(char));
                unGappedAlignGPU_stripedArr<<<1,NUM_CUDA_THREAD>>>(vectorCurr, vectorPrev, query[i], target, max, targetSize);
                hipDeviceSynchronize();

                hipMemcpy(maxHost, max, NUM_CUDA_THREAD * sizeof(char), hipMemcpyDeviceToHost);

                for(int ini=0; ini<NUM_CUDA_THREAD; ini++){
                    if(maxRes < maxHost[ini]) {
                        maxRes = maxHost[ini];
                        //printf("Report [%d] at query %d.%d\n", maxRes, i, ini);
                    }
                }

                char* vectorTmp = vectorPrev;
                vectorPrev = vectorCurr;
                vectorCurr = vectorTmp;
            }
        }
        GPUtime = clock() - begin_time;
        cout << "GPU result : " << (int) maxRes << "[striped with Array overhead]" << endl;
        cout << "Elapsed time : " << GPUtime / CLOCKS_PER_SEC << endl;

        hipFree(target); hipFree(vectorCurr); hipFree(vectorPrev); hipFree(max);
        delete[] maxHost;
    }
    cout << "Speed Ratio : G/C = " << GPUtime / CPUtime << endl;
    return 0;
}
